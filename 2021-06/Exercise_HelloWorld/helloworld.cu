/* Demonstration of a "Hello World" Cuda example in C */


#include <hip/hip_runtime.h>
#include <stdio.h>
#define NUM_BLOCKS 4
#define BLOCK_WIDTH 8


/* Function executed on device GPU */
__global__ 
void hello() 
{
  printf("\tHello from GPU: thread %d and block %d\n", threadIdx.x, blockIdx.x);

}

/* Main function, executed on host CPU */
int main() {

  /* print message from CPU */
  printf( "Hello Cuda From CPU!\n" );

  /* execute function on device */
  hello<<<NUM_BLOCKS, BLOCK_WIDTH>>>();

  /* wait until all threads finish their job */
  hipDeviceSynchronize();

  /* print message from CPU */
  printf( "Welcome back to CPU!\n" );

  return (0);
}
