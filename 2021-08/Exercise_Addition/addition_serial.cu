/* Demonstration of a simple kernel to add two integers  */
  

#include <hip/hip_runtime.h>
#include <stdio.h>
#define NUM_BLOCKS 1
#define BLOCK_WIDTH 1

/* Function executed on device GPU */
__global__
void add(int *a, int *b, int *c)
{
  // Note - we use pointers for the variables
  // add() runs on the device, so a, b and c must point to device memory
  *c = *a + *b;
}

/* Main function, executed on host CPU */
int main() {

  int a, b, c; // host copies of a, b, c
  int *d_a, *d_b, *d_c; // device copies of a, b, c
  int size = sizeof(int);

  // Allocate space for device copies of a, b, c
  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_c, size);

  // Setup input values:
  a = 3;
  b = 8;

  printf( "The parameters are: a = %d, b = %d \n", a, b);

  // Copy inputs to device:
  hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

  // Launch add() kernel on GPU:
  add<<<1,1>>>(d_a, d_b, d_c);

  // Copy result back to host:
  hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

  printf( "The result is: a + b  = %d \n", c);

  // Cleanup:
  hipFree(d_a); hipFree(d_b); hipFree(d_c);

  return 0;
}
