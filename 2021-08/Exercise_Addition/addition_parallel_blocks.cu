/* Demonstration of a simple kernel to add two integers  */
  

#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 10

/* Function to generate an array of random numbers (range 0-100) "arr" of size "m" */
void random_ints(int *arr, int m)
{
   int i;
   for (i = 0; i < m; ++i)
    arr[i] = rand() % 100;
}

/* Function executed on device GPU */
__global__
void add(int *a, int *b, int *c)
{
  // Note - we use pointers for the variables
  // add() runs on the device, so a, b and c must point to device memory
  c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
  printf("\tParameters on thread %d and block %d are: %d , %d \n Result: %d \n", threadIdx.x, blockIdx.x, a[blockIdx.x], b[blockIdx.x], c[blockIdx.x]);
  // By using blockIdx.x to index into the array, each block handles a different element of the array
}

/* Main function, executed on host CPU */
int main() {

  int *a, *b, *c; // host copies of a, b, c
  int *d_a, *d_b, *d_c; // device copies of a, b, c
  int size = N * sizeof(int);

  // Alloc space for device copies of a, b, c
  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_c, size);

  // Alloc space for host copies of a, b, c and setup input values (arrays):  
  a = (int *)malloc(size); random_ints(a, N);
  b = (int *)malloc(size); random_ints(b, N);
  c = (int *)malloc(size);
 
  // Copy inputs to device:
  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

  // Launch N copies of add() with add<<<N,1>>>(…);
  add<<<N,1>>>(d_a, d_b, d_c);

  // Copy result back to host:
  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

  // Cleanup:
  free(a); free(b); free(c);
  hipFree(d_a); hipFree(d_b); hipFree(d_c);

  return 0;
}
